#include "hip/hip_runtime.h"
#include "particle_system.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <random>

__global__ void assignParticlesToCells(Particle* particles, int* cell_start, int* cell_end, int* cell_particles, unsigned int num_particles, int GRID_SIZE, float simulation_box_width, float simulation_box_height);
__device__ void handleBoundaryCollision(Particle& p, float radius, float top_boundary, float bottom_boundary, float left_boundary, float right_boundary);
__global__ void updateParticlesKernel(Particle* particles, Particle* new_particles, float radius, unsigned int num_particles, float delta_time, float top_boundary, float bottom_boundary, float left_boundary, float right_boundary);
// ---

void updateParticlesCUDA(Particle* particles, Particle* new_particles, int* cell_start, int* cell_end, int* cell_particles, float radius, float GRID_SIZE, unsigned int num_particles, float delta_time, float simulation_box_width, float simulation_box_height);
// ---

ParticleSystem::ParticleSystem(unsigned int num_particles, int simulation_box_width, int simulation_box_height, UpdateMethod update_method) 
    : num_particles(num_particles), simulation_box_width(simulation_box_width), simulation_box_height(simulation_box_height), update_method(update_method)
{
    particles.resize(num_particles);
    
    initParticles();
    initGL();
    if (update_method == CUDA)
        initCUDA();

    shader = new Shader("./shaders/vertex_shader.glsl", "./shaders/fragment_shader.glsl");

    glEnable(GL_PROGRAM_POINT_SIZE);
    glEnable(GL_POINT_SPRITE);
}

ParticleSystem::~ParticleSystem()
{
    if (update_method == CUDA)
        hipGraphicsUnregisterResource(cudaVBO);

    glDeleteBuffers(1, &VBO);
    glDeleteVertexArrays(1, &VAO);
    delete shader;
    glDisable(GL_PROGRAM_POINT_SIZE);
    glDisable(GL_POINT_SPRITE);
}

void ParticleSystem::initParticles()
{
    // check if num_particles is sqaure.
    unsigned int side_length = static_cast<unsigned int>(std::sqrt(num_particles));
    if (side_length * side_length != num_particles)
    {
        std::cerr << "Number of particles must be a perfect square!" << std::endl;
        exit(EXIT_FAILURE);
    }

    float x_gl_res = 0.8f; // Change the variable name
    float y_gl_res = 0.8f; // Change the variable name

    float start_x = -x_gl_res * simulation_box_width / 2.0f;    // 1280 >> (-0.5 : 0.5) -> (-320 : 320)
    float start_y = -y_gl_res * simulation_box_height / 2.0f;   //  720 >> (-0.5 : 0.5) -> (-180 : 180)
    float step_x = (simulation_box_width / 2.0f) / side_length * (x_gl_res / 0.5f);
    float step_y = (simulation_box_height / 2.0f) / side_length * (y_gl_res / 0.5f);

    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(-2.0f, 2.0f);

    for (unsigned int i = 0; i < side_length; i++)
    {
        for (unsigned int j = 0; j < side_length; j++)
        {
            unsigned int index = i * side_length + j;
            float x = start_x + i * step_x;
            float y = start_y + j * step_y;

            float noise_x = distribution(generator);
            float noise_y = distribution(generator);

            particles[index].position = glm::vec3(x + noise_x, y + noise_y, 0.0f);
            particles[index].velocity = glm::vec3(0.0f, 0.0f, 0.0f);
            particles[index].life = 1.0f;
        }
    }

}

void ParticleSystem::initGL()
{
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, num_particles * sizeof(Particle), particles.data(), GL_DYNAMIC_DRAW);

    // vertex positions
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)0);

    glBindVertexArray(0);
}

void ParticleSystem::initCUDA()
{
    hipGraphicsGLRegisterBuffer(&cudaVBO, VBO, cudaGraphicsMapFlagsWriteDiscard);
    hipMalloc((void**)&d_new_particles, num_particles * sizeof(Particle));

    hipMalloc((void**)&d_cell_start, GRID_SIZE * GRID_SIZE * sizeof(int));
    hipMalloc((void**)&d_cell_end, GRID_SIZE * GRID_SIZE * sizeof(int));
    hipMalloc((void**)&d_cell_particles, num_particles * sizeof(int));
}

void ParticleSystem::update(float delta_time)
{
    if(update_method == CUDA)
        updateCUDA(delta_time);
    else
        updateCPU(delta_time);
}

void ParticleSystem::updateCPU(float delta_time)
{
    // update something
    glm::vec3 gravity(0.0f, -9.8f, 0.0f);

    for (unsigned int i = 0; i < num_particles; i++)
    {
        for (unsigned int j = i + 1; j < num_particles; j++)
        {
            glm::vec3 delta = particles[j].position - particles[i].position;
            float distance = glm::length(delta);
            
            // collision detected
            if (distance < 2 * radius)
            {
                glm::vec3 normal = glm::normalize(delta);
                glm::vec3 relative_velocity = particles[j].velocity - particles[i].velocity;
                float velocity_along_normal = glm::dot(relative_velocity, normal);
                
                // collision is detected but particles are moving apart
                if(velocity_along_normal > 0)
                    continue;

                float restitution = 0.9f;
                float impulse_magnitude = -1.0f * (1 + restitution) * velocity_along_normal / 2.0f;

                glm::vec3 impulse = impulse_magnitude * normal;
                particles[i].velocity -= impulse;
                particles[j].velocity += impulse;

                float overlap = 2 * radius - distance;
                particles[i].position -= normal * overlap / 2.0f;
                particles[j].position += normal * overlap / 2.0f;
            }
        }
    }

    float top_boundary      = 1.0f * simulation_box_height / 2.0f;
    float bottom_boundary   = -1.0f * simulation_box_height / 2.0f;
    float left_boundary     = -1.0f * simulation_box_width / 2.0f;
    float right_boundary    = 1.0f * simulation_box_width / 2.0f;
        
    for (auto& p : particles)
    {
        p.velocity += gravity * delta_time;
        p.position += p.velocity * delta_time;
        p.life -= delta_time;

        if (p.position.y + radius > top_boundary)
        {
            p.position.y = top_boundary - radius;
            p.velocity.y *= -0.9f;
        }
        if (p.position.y - radius < bottom_boundary)
        {
            p.position.y = bottom_boundary + radius;
            p.velocity.y *= -0.9f;
        }        
        if (p.position.x - radius < left_boundary)
        {
            p.position.x = left_boundary + radius;
            p.velocity.x *= -0.9f;
        }
        if (p.position.x + radius > right_boundary)
        {
            p.position.x = right_boundary - radius;
            p.velocity.x *= -0.9f;
        }
    }

    // update vbo
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferSubData(GL_ARRAY_BUFFER, 0, num_particles * sizeof(Particle), particles.data());
}

void ParticleSystem::updateCUDA(float delta_time)
{
    Particle* d_particles;
    size_t num_bytes;

    // map vbo
    hipGraphicsMapResources(1, &cudaVBO, 0);
    hipGraphicsResourceGetMappedPointer((void**)&d_particles, &num_bytes, cudaVBO);

    // update something and vbo
    updateParticlesCUDA(d_particles, d_new_particles, d_cell_start, d_cell_end, d_cell_particles, radius, GRID_SIZE, num_particles, delta_time, simulation_box_width, simulation_box_height);

    // unmap vbo
    hipGraphicsUnmapResources(1, &cudaVBO, 0);
}

void ParticleSystem::render(float time)
{
    shader->use();
    shader->setInt("simulation_box_width", simulation_box_width);
    shader->setInt("simulation_box_height", simulation_box_height);
    shader->setFloat("time", time);

    glBindVertexArray(VAO);
    glDrawArrays(GL_POINTS, 0, num_particles);
    glBindVertexArray(0);
}

__global__ void assignParticlesToCells(Particle* particles, int* cell_start, int* cell_end, int* cell_particles, unsigned int num_particles, int GRID_SIZE, float simulation_box_width, float simulation_box_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= num_particles)
        return;

    Particle& p = particles[idx];
    int cellX = int((p.position.x + simulation_box_width / 2.0f) / GRID_SIZE);
    int cellY = int((p.position.y + simulation_box_height / 2.0f) / GRID_SIZE);

    int cell_idx = cellY * GRID_SIZE + cellX;

    atomicMin(&cell_start[cell_idx], idx);
    atomicMax(&cell_start[cell_idx], idx);

    cell_particles[idx] = cell_idx;
}

__device__ void handleBoundaryCollision(Particle& p, float radius, float top_boundary, float bottom_boundary, float left_boundary, float right_boundary)
{
    if (p.position.y + radius > top_boundary)
    {
        p.position.y = top_boundary - radius;
        p.velocity.y *= -0.9f;
    }
    if (p.position.y - radius < bottom_boundary)
    {
        p.position.y = bottom_boundary + radius;
        p.velocity.y *= -0.9f;
    }
    if (p.position.x + radius > right_boundary)
    {
        p.position.x = right_boundary - radius;
        p.velocity.x *= -0.9f;
    }
    if (p.position.x - radius < left_boundary)
    {
        p.position.x = left_boundary + radius;
        p.velocity.x *= -0.9f;
    }
}

__global__ void updateParticlesKernel(Particle* particles, Particle* new_particles, float radius, unsigned int num_particles, float delta_time, float top_boundary, float bottom_boundary, float left_boundary, float right_boundary)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_particles) return;

    new_particles[idx] = particles[idx];
    Particle& p = particles[idx];
    
    // Handle particle collisions
    for (unsigned int j = 0; j < num_particles; j++)
    {
        if (j == idx)
            continue;
        
        Particle& q = particles[j];
        glm::vec3 delta = q.position - p.position;
        float distance = glm::length(delta);

        // collision detected
        if (distance < 2 * radius)
        {
            glm::vec3 normal = glm::normalize(delta);
            glm::vec3 relative_velocity = q.velocity - p.velocity;
            float velocity_along_normal = glm::dot(relative_velocity, normal);
            if (velocity_along_normal > 0)
                continue;
            
            float restitution = 1.0f;
            float impulse_magnitude = -1.0f * (1.0f + restitution) * velocity_along_normal / 2.0f;

            glm::vec3 impulse = impulse_magnitude * normal;
            new_particles[idx].velocity -= impulse;
            new_particles[idx].velocity *= 0.99f;
            
            float overlap = 2 * radius - distance;
            new_particles[idx].position -= normal * overlap / 2.0f;
        }
    }

    // Apply gravity and wall collision
    glm::vec3 gravity(0.0f, -0.98f * 0.5f, 0.0f);
    new_particles[idx].velocity += gravity * delta_time;
    new_particles[idx].position += new_particles[idx].velocity * delta_time;
    new_particles[idx].life -= delta_time;

    handleBoundaryCollision(new_particles[idx], radius, top_boundary, bottom_boundary, left_boundary, right_boundary);
}

void updateParticlesCUDA(Particle* particles, 
                        Particle* new_particles,
                        int* cell_start, 
                        int* cell_end, 
                        int* cell_particles, 
                        float radius, 
                        float GRID_SIZE,
                        unsigned int num_particles, 
                        float delta_time, 
                        float simulation_box_width, 
                        float simulation_box_height)
{
    // Boundary
    float top_boundary      = 1.0f * simulation_box_height / 2.0f;
    float bottom_boundary   = -1.0f * simulation_box_height / 2.0f;
    float left_boundary     = -1.0f * simulation_box_width / 2.0f;
    float right_boundary    = 1.0f * simulation_box_width / 2.0f;

    hipMemset(cell_start, -1, GRID_SIZE * GRID_SIZE * sizeof(int));
    hipMemset(cell_end, -1, GRID_SIZE * GRID_SIZE * sizeof(int));

    int threadsPerBlock = 256;
    int blocksPerGrid = (num_particles + threadsPerBlock - 1) / threadsPerBlock;

    assignParticlesToCells<<<blocksPerGrid, threadsPerBlock>>>(particles, cell_start, cell_end, cell_particles, num_particles, GRID_SIZE, simulation_box_width, simulation_box_height);
    hipDeviceSynchronize();

    updateParticlesKernel<<<blocksPerGrid, threadsPerBlock>>>(particles, new_particles, radius, num_particles, delta_time, top_boundary, bottom_boundary, left_boundary, right_boundary);
    hipMemcpy(particles, new_particles, num_particles * sizeof(Particle), hipMemcpyDeviceToDevice);
}